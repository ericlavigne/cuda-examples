
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void square(float* d_out, float* d_in) {
  int idx = threadIdx.x;
  float f = d_in[idx];
  d_out[idx] = f * f;
}

int main(int argc, char** argv) {
  const int ARRAY_SIZE = 64;
  const int ARRAY_BYTES = ARRAY_SIZE * sizeof(float);

  // Generate input array on host
  float h_in[ARRAY_SIZE];
  for(int i = 0; i < ARRAY_SIZE; i++) {
    h_in[i] = float(i);
  }
  // Generate output array on host
  float h_out[ARRAY_SIZE];

  // Declare GPU (device) memory pointers
  float * d_in;
  float * d_out;

  // Allocate GPU memory
  hipMalloc((void**) &d_in, ARRAY_BYTES);
  hipMalloc((void**) &d_out, ARRAY_BYTES);

  // Transfer input array to GPU
  hipMemcpy(d_in, h_in, ARRAY_BYTES, hipMemcpyHostToDevice);

  // Launch the kernel with one block of ARRAY_SIZE threads
  square<<<1, ARRAY_SIZE>>>(d_out, d_in);

  // Copy result back to the CPU
  hipMemcpy(h_out, d_out, ARRAY_BYTES, hipMemcpyDeviceToHost);

  // Print out the result
  for(int i = 0; i < ARRAY_SIZE; i++) {
    printf("%f", h_out[i]);
    printf(((i % 4) == 3) ? "\n" : "\t");
  }

  // Free GPU memory allocation
  hipFree(d_in);
  hipFree(d_out);

  return 0;
}

